#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include <atomic>
#include <chrono>
#include <iostream>
#include <sstream>
#include <string>
#include <thread>

#include "common/command_line.h"
#include "common/meta.h"
#include "common/time_measurer.h"
#include "executor/pipeline_executor.h"
#include "graph/cpu_graph.h"
#include "query/clique.h"
#include "query/graphlet.h"
#include "query/pattern.h"
#include "query/plan.h"
#include "query/query.h"
#include "rigtorp/MPMCQueue.h"
#include "view/view_bin.h"
#include "view/view_bin_buffer.h"
#include "view/view_bin_holder.h"
#include "view/view_bin_manager.h"

int main(int argc, char** argv) {
  if (argc == 1) {
    return -1;
  }

  // parse command line
  CommandLine cmd(argc, argv);
  std::string filename = cmd.GetOptionValue("-f", "./data/com-friendster.ungraph.txt.bin");
  double mem = cmd.GetOptionDoubleValue("-m", 10);
  int pattern_type = cmd.GetOptionIntValue("-p", P1);
  int k_cliques = cmd.GetOptionIntValue("-kc", 0);
  int k_motifs = cmd.GetOptionIntValue("-km", 0);
  int thread_num = cmd.GetOptionIntValue("-t", 1);
  int queue_size = cmd.GetOptionIntValue("-qs", 1);
  int producers_num = cmd.GetOptionIntValue("-pn", 1);
  int consumers_num = cmd.GetOptionIntValue("-cn", 1);
  int do_match = cmd.GetOptionIntValue("-dm", 1);
  int do_filter = cmd.GetOptionIntValue("-df", 1);
  int do_reorder = cmd.GetOptionIntValue("-dr", 0);
  int do_split = cmd.GetOptionIntValue("-ds", 0);
  int do_split_times = cmd.GetOptionIntValue("-dst", 1);

  // Check if enough gpus for consumers
  int gpu_count = 0;
  hipGetDeviceCount(&gpu_count);
  assert(consumers_num <= gpu_count);

  // Check if producers_num >= consumers_num && queue_size >= consumers_num
  assert(queue_size >= consumers_num);
  std::cout << "m: " << mem << " t: " << thread_num << " qs: " << queue_size << " pn: " << producers_num << " cn: " << consumers_num << std::endl;

  // Get Query
  std::vector<Query*> queries;
  if (k_cliques != 0) {
    Query* query = new Clique(k_cliques);
    query->Print();
    queries.push_back(query);
  } else if (k_motifs != 0) {
    Graphlet* graphlet = new Graphlet(k_motifs);
    queries = std::move(graphlet->GetQueries());
  } else {
    // one pattern
    Query* query = new Pattern((PresetPatternType)pattern_type, false);
    query->Print();
    queries.push_back(query);
  }

  // Query to Plan
  std::vector<Plan*> plans;
  int hop = 0;
  size_t root_degree = do_filter ? kMaxsize_t : 0;
  for (auto& query : queries) {
    Plan* plan = new Plan(query);
    plan->Optimize();
    plan->Print();
    plans.push_back(plan);
    if (plan->GetHop() > hop)
      hop = plan->GetHop();
    if (plan->GetRootDegree() < root_degree)
      root_degree = plan->GetRootDegree();
  }

  // Load data graph
  Graph* graph = new Graph(filename, false);

  // Load view bins partition
  ViewBinManager* vbm = new ViewBinManager(graph, hop, thread_num);
  std::string partition_file = filename + "." + std::to_string(hop) + ".hop.vbmap";
  vbm->LoadViewBinPartition(partition_file);
  if (do_reorder)
    vbm->Reorder();
  if (do_split)
    vbm->Split(consumers_num, do_split_times);
  size_t max_partitioned_sources_num = vbm->GetMaxPartitionedSourcesNum();
  size_t max_view_bin_size = mem * 1000 * 1000 * 1000;  // already the total size, no need to multiply 4 bytes

  // Initialize view bins buffer
  ViewBinBuffer view_bin_buffer(queue_size, graph->GetVertexCount(), max_view_bin_size);

  rigtorp::MPMCQueue<int> assigned_queue(queue_size);
  rigtorp::MPMCQueue<int> released_queue(queue_size);
  // fill up the queue first
  for (int i = 0; i < queue_size; i++)
    released_queue.push(i);

  std::vector<std::thread> threads;
  std::atomic<int> view_bin_pool_index{0};
  std::atomic<int> num_finished_producers{0};
  auto& view_bin_pool = vbm->GetViewBinPool();
  // Multiple Producers
  for (int p = 0; p < producers_num; p++)
    threads.push_back(std::thread([p, queue_size, root_degree, &num_finished_producers, &view_bin_pool_index, &view_bin_pool, &view_bin_buffer, &assigned_queue, &released_queue]() {
      for (int view_bin_id{}; (view_bin_id = view_bin_pool_index++) < view_bin_pool.size();) {
        int holder_id = -1;
        released_queue.pop(holder_id);
        // std::this_thread::sleep_for(std::chrono::milliseconds(1000));
        std::stringstream a;
        a << "producer " << p << " gets " << holder_id << std::endl;
        std::cout << a.str();
        view_bin_pool[view_bin_id]->Materialize(view_bin_buffer.GetViewBinHolder(holder_id), root_degree);
        std::stringstream b;
        b << "producer " << p << " produces " << view_bin_id << " in " << holder_id << std::endl;
        std::cout << b.str();
        assigned_queue.push(holder_id);
      }
      // signal to stop the consumers
      num_finished_producers++;
    }));

  std::vector<std::vector<uintC>> counts(consumers_num);
  for (int c = 0; c < consumers_num; c++)
    counts[c].resize(plans.size(), 0);

  // Multiple Consumers
  for (int c = 0; c < consumers_num; c++)
    threads.push_back(std::thread([c, producers_num, do_match, max_partitioned_sources_num, &graph, &plans, &num_finished_producers, &view_bin_buffer, &assigned_queue, &released_queue, &counts]() {
      PipelineExecutor* executor = new PipelineExecutor(c, graph, plans, max_partitioned_sources_num);
      while (true) {
        int holder_id = -1;
        if (num_finished_producers == producers_num) {  // stop condition
          // std::this_thread::sleep_for(std::chrono::milliseconds(10));  // avoid busy waiting
          if (!assigned_queue.try_pop(holder_id))  // try pop again and exit if no more items
            break;
        } else if (!assigned_queue.try_pop(holder_id)) {  // try pop if empty
          // std::this_thread::sleep_for(std::chrono::milliseconds(10));  // avoid busy waiting
          continue;
        }
        // std::this_thread::sleep_for(std::chrono::milliseconds(2000));
        std::stringstream a;
        a << "consumer " << c << " gets " << holder_id << std::endl;
        std::cout << a.str();
        executor->Transfer(view_bin_buffer.GetViewBinHolder(holder_id));
        std::stringstream b;
        b << "consumer " << c << " completes transfer " << holder_id << std::endl;
        std::cout << b.str();
        // can release after transfer
        released_queue.push(holder_id);
        executor->Match(do_match);
        std::stringstream d;
        d << "consumer " << c << " completes match " << holder_id << std::endl;
        std::cout << d.str();
        // executor->PrintTotalCounts();
      }
      counts[c] = executor->GetTotalCounts();
      std::cout << "consumer " << c << " stops" << std::endl;
    }));

  TimeMeasurer timer;
  timer.StartTimer();

#if defined(NVPROFILE)
  hipProfilerStart();
#endif

  for (auto& t : threads)
    t.join();

#if defined(NVPROFILE)
  hipProfilerStop();
#endif

  timer.EndTimer();
  timer.PrintElapsedMicroSeconds("total time");

  // Total count
  std::vector<uintC> total_counts(plans.size(), 0);
  for (int c = 0; c < consumers_num; c++)
    for (int i = 0; i < plans.size(); i++)
      total_counts[i] += counts[c][i];

  for (int i = 0; i < plans.size(); i++)
    std::cout << "total count for query " << i << ": " << total_counts[i] << std::endl;

  return 0;
}
